#include "hip/hip_runtime.h"
/**
 * @file gpu_benchmark.cu
 * benchmark for GPU
 * author: Linlin Chen
 * lchen96@hawk.iit.edu
 * 
 */

#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <sys/time.h>
#include <unistd.h>
#include <gpu_benchmark.h>

using namespace std;


/**
 * user manual, help user how to give input
 */
void helper (char *arg) {
	cout<<arg<<": GPU benchmark tool, Version 0.0.1 (lchen96@hawk.iit.edu)"<<endl<<endl;
	cout<<"usage:\t"<<arg<<" [-h] [-b] [-o <opearation>] [-l <option>] [-r <option>]"<<endl;
	cout<<"Arguments:"<<endl;
	cout<<"\t-h\tlist available commands"<<endl;
	cout<<"\t-b\ttest GPU bandwidth only"<<endl;
	cout<<"\t-o\toperation type, double=0, integer=1, half=2, quarter=3"<<endl;
	cout<<"\t-l\tnumber of loops[default = 8e^8]"<<endl;
	cout<<"\t-r\tnumber of repeated benchmark tests[default = 1]"<<endl;
	cout<<endl;
}






int main (int argc, char * argv[]) {
	/*
	use getopt function to parse the input arguments
	 */
	int c;
	int flag;
	// only accepts "hfitlr" arguments, where for "tlr" the argument can be optional
	while ((c = getopt (argc, argv, ":hbo::l::r::")) != -1) 
		switch (c){
			case 'h':
				helper();
				exit(0);
				break;
			case 'b':
				testBandwidth = true;
				break;
			case 'o':
				flag = stoi(optarg);
				if (flag == 0)
					op_type = FLOP;
				else if (flag == 1)
					op_type = IOP;
				else if	(flag == 2)
					op_type = HOP;
				else if (flag == 3)
					op_type = QOP;
				else {
					cerr<<"-o: option type can only be 0, 1, 2, 3!\n"<<endl;
					helper(argv[0]);
					exit(1);
				}
				break;
			case 'l':
				loop_num = (long) atof(optarg);
			case 'r':
				repeat_num = stoi(optarg);
			case ':':
				if (optopt == 'o')
					op_type = FLOP;
				else if (optopt == 'l') 
					loop_num = DEFAULTLOOP;
				else if (optopt == 'r') 
					repeat_num = 1;
				break;
			case '?':
			default:
				cerr<<"Invalid arguments!"<<endl;
				helper(argv[0]);
				abort();
		}

		//output user's setting information for benchmark
		cout<<"\nThe benchmarking begins with:"
			<<"\n\tOperation:\t\t"<<op[op_type]
			<<"\n\t#Loop:\t\t\t"<<loop_num
			<<"\n\t#Iteration:\t\t"<<repeat_num<<endl<<endl<<endl;


		int config[2] = {0};
		core_num = computeConfiguration(config);	//get the number of GPU cores

		float *runtime = new float[repeat_num];
		cout<<"GPU Info:\n\tGPU: "<<dev.name<<"\n\tCore Num: "<<core_num
		if (testBandwidth) {
			cout<<"GPU\tBlockSize\tDataSize\tBandwidth(MB/s)\tTime"<<endl;
		} else {
			if (op_type == FLOP)
				cout<<"GPU\tOptType\t#Operation\tGFLOPS\tTime"<<endl;
			else if (op_type == IOP)
				cout<<"GPU\tOptType\t#Operation\tGIOPS\tTime"<<endl;
			else if (op_type == HOP)
				cout<<"GPU\tOptType\t#Operation\tGHOPS\tTime"<<endl;
			else if (op_type == QOP)
				cout<<"GPU\tOptType\t#Operation\tGQOPS\tTime"<<endl;
		}
		for (int i = 0; i < repeat_num; i++) {
			//runtime[i] = gpu_benchmark();
			if (testBandwidth) {
				runtime[i] = gpu_bdwth_benchmark();

				cout<<"#Iter "<<i<<"\t"<<BYTE_IN_MB(BLOCKSIZE)<<"MB\t"
				<<BYTE_IN_MB(BLOCKSIZE*block_num)<<"MB\t"
				<<BYTE_IN_MB(BLOCKSIZE*block_num)/runtime[i]<<"MB/s\t"<<runtime[i]<<"s"<<endl;
			} else {
				runtime[i] = gpu_thrpt_benchmark();

				cout<<"#Iter "<<i<<"\t"<<op[op_type]<<"\t"<<loop_num<<"\t"
					<<(loop_num * core_num) / runtime[i] / 1e9<<"\t"<<runtime[i]<<"s"<<endl;
			}
		}


}


double gpu_bdwth_benchmark () {
	hipEvent_t start, stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);

  	
}

double gpu_thrpt_benchmark () {

}




//kernel function for double precision operation
__global__
void DFL_Ops() {


}

//kernel function for integer operation
__global__
void Int_Ops() {

}


//kernel function for half-precision
__global__
void H_Ops() {

}

//kernel function for quater-precision
__global__
void Q_Ops() {

}
